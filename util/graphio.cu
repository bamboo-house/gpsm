#include "hip/hip_runtime.h"
#include "graphio.h"

namespace gpsm {
namespace graphio {
	//---------------------------------------------------------------------------
	void getLabels(GPGraph* graph) { // get statistics of node labels
		if (graph->numNodes > 0) {
			graph->numLabels = 0;

			FOR_LIMIT(i, graph->numNodes) graph->numLabels = std::max(graph->numLabels, graph->nodeLabels[i]);
			graph->numLabels++;

			graph->labelSizes = (int*)malloc(graph->numLabels * sizeof(int));
			CHECK_POINTER(graph->labelSizes);

			FOR_LIMIT(i, graph->numLabels) graph->labelSizes[i] = 0;
			FOR_LIMIT(i, graph->numNodes) graph->labelSizes[graph->nodeLabels[i]]++;

			graph->maxLabelSize = 0;
			FOR_LIMIT(i, graph->numLabels)graph->maxLabelSize = std::max(graph->maxLabelSize, graph->labelSizes[i]);
		}
	}
	//---------------------------------------------------------------------------
	void getInEdges(GPGraph* graph) { // get information about incoming nodes and edges
		if (graph->numNodes > 0) {
			graph->inOffsets = (int*)malloc((graph->numNodes + 1) * sizeof(int));
			CHECK_POINTER(graph->inOffsets);

			graph->inEdges = (int*)malloc(graph->numEdges * sizeof(int));
			CHECK_POINTER(graph->inEdges);

			// calculate in-offsets of nodes
			FOR_LIMIT(i, graph->numNodes)
				FOR_RANGE(j, graph->outOffsets[i], graph->outOffsets[i + 1]) {
				int node2 = graph->outEdges[j];
				graph->inOffsets[node2]++;
			}
			graph->inOffsets[graph->numNodes] = 0;

			// prefix sum
			FOR_LIMIT(i, graph->numNodes) graph->inOffsets[i + 1] += graph->inOffsets[i];

			// calculate in-endpoints
			FOR_LIMIT_REV(i, graph->numNodes)
				FOR_RANGE(j, graph->outOffsets[i], graph->outOffsets[i + 1]) {
				int node = graph->outEdges[j];
				int offset = --graph->inOffsets[node];
				graph->inEdges[offset] = i;
			}
		}
	}
	//---------------------------------------------------------------------------
	bool readBinary(GPGraph* graph, char* fileName, bool debug) {
		FILE* fp = fopen(fileName, "rb");
		if (fp == NULL) return false;

		if (debug) {
			printf("Loading graph from binary file ...\n");
		}

		// read statistics
		fread(&graph->numNodes, sizeof(int), 1, fp);

		// read contents
		graph->nodeLabels = (int*)malloc(graph->numNodes * sizeof(int));
		CHECK_POINTER(graph->nodeLabels);

		graph->outOffsets = (int*)malloc((graph->numNodes + 1) * sizeof(int));
		CHECK_POINTER(graph->outOffsets);

		fread(graph->nodeLabels, sizeof(int), graph->numNodes, fp);
		fread(graph->outOffsets, sizeof(int), graph->numNodes + 1, fp);

		graph->numEdges = graph->outOffsets[graph->numNodes];
		graph->outEdges = (int*)malloc(graph->numEdges * sizeof(int));
		CHECK_POINTER(graph->outEdges);

		fread(graph->outEdges, sizeof(int), graph->numEdges, fp);

		// close file
		fclose(fp);

		// calculate label statistics
		getLabels(graph);

		// calculate incoming information
		getInEdges(graph);

		return true;
	}
	//---------------------------------------------------------------------------
	bool writeBinary(GPGraph* graph, char* fileName, bool debug) {
		FILE* fp = fopen(fileName, "wb");
		if (fp == NULL) return false;

		if (debug) {
			printf("Writing graph to binary file ...\n");
		}

		// write statistics
		fwrite(&graph->numNodes, sizeof(int), 1, fp);

		// write contents
		fwrite(graph->nodeLabels, sizeof(int), graph->numNodes, fp);
		fwrite(graph->outOffsets, sizeof(int), graph->numNodes + 1, fp);
		fwrite(graph->outEdges, sizeof(int), graph->numEdges, fp);

		// close file
		fclose(fp);

		return true;
	}
	//---------------------------------------------------------------------------
	bool readStatistics(GPGraph* graph, char* fileName, bool debug) { // read graph statistics from text format
		std::ifstream in(fileName);

		if (in.is_open()) {
			if (debug) {
				printf("Reading graph statistics from text file ...\n");
			}

			std::string line;
			while (getline(in, line)) {
				if (line[0] == 'v') graph->numNodes++;
				else if (line[0] == 'e') graph->numEdges++;
			}

			in.close();
			return true;
		}

		return false;
	}
	//---------------------------------------------------------------------------
	bool readNodes(GPGraph* graph, char* fileName, bool debug) { // read node information from text format
		std::ifstream in(fileName);

		if (in.is_open()) {
			if (debug) {
				printf("Reading node information from text file ...\n");
			}

			std::string line;
			while (getline(in, line)) {
				if (line[0] == 'v') { // get node information
					std::istringstream iss(line);

					std::string type;
					int node;
					int label;

					if (!(iss >> type >> node >> label)) {
						printf("Graph format errors!...\n");
						return false;
					} // error

					if (node >= graph->numNodes) {
						printf("Graph format errors!...\n");
						return false;
					} // error

					graph->nodeLabels[node] = label;
				}
				else if (line[0] == 'e') { // get edge information

					std::istringstream iss(line);
					std::string type;
					int node1;
					int node2;

					if (!(iss >> type >> node1 >> node2)) {
						printf("Graph format errors!...\n");
						return false;
					} // error

					if (node1 >= graph->numNodes || node2 >= graph->numNodes) {
						printf("Graph format errors!...\n");
						return false;
					} // error

					graph->outOffsets[node1]++;
					graph->inOffsets[node2]++;
				}
			}

			in.close();

			return true;
		}

		return false;
	}
	//---------------------------------------------------------------------------
	bool readEdges(GPGraph* graph, char* fileName, bool debug) { // get edge information from text format
		std::ifstream in(fileName);

		if (in.is_open()) {
			if (debug) {
				printf("Reading graph contents from text file ...\n");
			}

			// prefix sum
			FOR_LIMIT(i, graph->numNodes) {
				graph->inOffsets[i + 1] += graph->inOffsets[i];
				graph->outOffsets[i + 1] += graph->outOffsets[i];
			}

			std::string line;
			while (getline(in, line)) {
				if (line[0] == 'e') { // get edge information
					std::istringstream iss(line);
					std::string type;
					int node1;
					int node2;

					if (!(iss >> type >> node1 >> node2)) {
						printf("Graph format errors!...\n");
						return false;
					} // error

					graph->outEdges[--graph->outOffsets[node1]] = node2;
					graph->inEdges[--graph->inOffsets[node2]] = node1;
				}
			}

			in.close();
			return true;
		}

		return false;
	}
	//---------------------------------------------------------------------------
	bool readText(GPGraph* graph, char* fileName, bool debug) {

		if (readStatistics(graph, fileName, debug) == false) return false;

		// init node and edge arrays
		graph->nodeLabels = (int*)malloc(graph->numNodes * sizeof(int));
		CHECK_POINTER(graph->nodeLabels);

		graph->outOffsets = (int*)malloc((graph->numNodes + 1) * sizeof(int));
		CHECK_POINTER(graph->outOffsets);
		FILL(graph->outOffsets, graph->numNodes + 1, 0);

		graph->inOffsets = (int*)malloc((graph->numNodes + 1) * sizeof(int));
		CHECK_POINTER(graph->inOffsets);
		FILL(graph->inOffsets, graph->numNodes + 1, 0);

		graph->outEdges = (int*)malloc(graph->numEdges * sizeof(int));
		CHECK_POINTER(graph->outEdges);

		graph->inEdges = (int*)malloc(graph->numEdges * sizeof(int));
		CHECK_POINTER(graph->inEdges);

		// get node information
		if (readNodes(graph, fileName, debug) == false) return false;

		// get label information
		getLabels(graph);

		// get edge information
		return readEdges(graph, fileName, debug);
	}
	//---------------------------------------------------------------------------
	bool writeText(GPGraph* graph, char* fileName, bool debug) { // read graph data from binary file
		std::ofstream out(fileName);
		if (out.is_open()) {
			FOR_LIMIT(i, graph->numNodes) out << "v " << i << " " << graph->nodeLabels[i] << std::endl;

			FOR_LIMIT(i, graph->numNodes)
				FOR_RANGE(j, graph->outOffsets[i], graph->outOffsets[i + 1])
				out << "e " << i << " " << graph->outEdges[j] << std::endl;

			out.close();
			return true;
		}

		return false;
	}
	//---------------------------------------------------------------------------
	bool copy(GPGraph* dest, GPGraph* src, CopyType type) {
		if ((type == CopyType::HOST_TO_DEVICE || type == CopyType::HOST_TO_HOST)
			&& src->dataPos != DataPosition::MEM) return NULL;

		if (type == CopyType::DEVICE_TO_HOST && src->dataPos != DataPosition::GPU) return NULL;

		dest->numNodes = src->numNodes;
		dest->numEdges = src->numEdges;
		dest->numLabels = src->numLabels;
		dest->maxLabelSize = src->maxLabelSize;

		switch (type)
		{
		case HOST_TO_DEVICE:
			dest->dataPos = DataPosition::GPU;

			CUDA_SAFE_CALL(hipMalloc(&dest->nodeLabels, src->numNodes * sizeof(int)));
			CUDA_SAFE_CALL(hipMalloc(&dest->outOffsets, (src->numNodes + 1) * sizeof(int)));
			CUDA_SAFE_CALL(hipMalloc(&dest->inOffsets, (src->numNodes + 1) * sizeof(int)));
			CUDA_SAFE_CALL(hipMalloc(&dest->outEdges, src->numEdges * sizeof(int)));
			CUDA_SAFE_CALL(hipMalloc(&dest->inEdges, src->numEdges * sizeof(int)));
			CUDA_SAFE_CALL(hipMalloc(&dest->labelSizes, src->numLabels * sizeof(int)));

			CUDA_SAFE_CALL(hipMemcpy(dest->nodeLabels, src->nodeLabels, src->numNodes * sizeof(int),
				hipMemcpyHostToDevice));

			CUDA_SAFE_CALL(hipMemcpy(dest->outOffsets, src->outOffsets, (src->numNodes + 1) * sizeof(int),
				hipMemcpyHostToDevice));

			CUDA_SAFE_CALL(hipMemcpy(dest->inOffsets, src->inOffsets, (src->numNodes + 1) * sizeof(int),
				hipMemcpyHostToDevice));

			CUDA_SAFE_CALL(hipMemcpy(dest->outEdges, src->outEdges, src->numEdges * sizeof(int),
				hipMemcpyHostToDevice));

			CUDA_SAFE_CALL(hipMemcpy(dest->inEdges, src->inEdges, src->numEdges * sizeof(int),
				hipMemcpyHostToDevice));

			CUDA_SAFE_CALL(hipMemcpy(dest->labelSizes, src->labelSizes, src->numLabels * sizeof(int),
				hipMemcpyHostToDevice));

			return true;
		case HOST_TO_HOST:
			dest->dataPos = DataPosition::MEM;

			dest->nodeLabels = (int*)malloc(src->numNodes * sizeof(int));
			CHECK_POINTER(dest->nodeLabels);

			dest->outOffsets = (int*)malloc((src->numNodes + 1) * sizeof(int));
			CHECK_POINTER(dest->outOffsets);

			dest->inOffsets = (int*)malloc((src->numNodes + 1) * sizeof(int));
			CHECK_POINTER(dest->inOffsets);

			dest->outEdges = (int*)malloc(src->numEdges * sizeof(int));
			CHECK_POINTER(dest->outEdges);

			dest->inEdges = (int*)malloc(src->numEdges * sizeof(int));
			CHECK_POINTER(dest->inEdges);

			dest->labelSizes = (int*)malloc(src->numLabels * sizeof(int));
			CHECK_POINTER(dest->labelSizes);

			memcpy(dest->nodeLabels, src->nodeLabels, src->numNodes * sizeof(int));
			memcpy(dest->outOffsets, src->outOffsets, (src->numNodes + 1) * sizeof(int));
			memcpy(dest->inOffsets, src->inOffsets, (src->numNodes + 1) * sizeof(int));
			memcpy(dest->outEdges, src->outEdges, src->numEdges * sizeof(int));
			memcpy(dest->inEdges, src->inEdges, src->numEdges * sizeof(int));
			memcpy(dest->labelSizes, src->labelSizes, src->numLabels * sizeof(int));
			
			return true;
		case DEVICE_TO_HOST:
			break;
		default:
			break;
		}

		return false;
	}
}}